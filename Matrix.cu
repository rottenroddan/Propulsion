#include "hip/hip_runtime.h"
//
// Created by steve on 7/31/2020.
//
#include "Propulsion.cuh"

template<typename type>
void Propulsion::Matrix<type>::generateMatrix(MatrixInitVal miv, MatrixInitType mit, type customVal, type *array)
{
    switch(miv)
    {
        // Case if requested a zero matrix
        case MatrixInitVal::zero:
            this->M = generateZeroMatrixArray(this->rows * this->cols);
            break;

        // Case if requested a null matrix.
        // Same as zero matrix but NULL instead, in case of objects used.
        case MatrixInitVal::null:
            this->M = generateNullMatrixArray(this->rows * this->cols);
            break;

        // Other value than zero.
        default:
            type val;
            if(miv == MatrixInitVal::ones)
            {
                val = 1; // Enum is one
            }
            else if(miv == MatrixInitVal::twos)
            {
                val = 2; // Enum is two
            }
            else if(miv == MatrixInitVal::custom)
            {
                val = customVal;    // Enum is a custom val
            }


            switch(mit)
            {
                // Default matrix, filled with custom val or a provided array of type now.
                case MatrixInitType::def:
                    this->M = generateDefaultMatrixArray(this->rows * this->cols, val, array);
                    break;
                // Diagonal Matrix to generate.
                case MatrixInitType::diagonal:
                    this->M = generateDiagonalMatrixArray(this->rows * this->cols, val, array);
                    break;
            }
    }
}

template<typename type>
void Propulsion::Matrix<type>::createPagedMemory()
{
    type* _pagedArr;
    _pagedArr = new type[this->totalSize];
    this->M = std::unique_ptr<type[], void(*)(type*)>(_pagedArr, freePagedMemory);
}

template<typename type>
void Propulsion::Matrix<type>::createPinnedMemory()
{
    type* _pinnedArr;
    gpuErrchk(hipHostMalloc((void**)&_pinnedArr, this->totalSize * sizeof(type)));
    this->M = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);
}


template<typename type>
Propulsion::Matrix<type>::Matrix() : M(nullptr, freePagedMemory)
{
    this->rows = 1;
    this->cols = 1;
    this->totalSize = 1;
    this->memoryType = MatrixMemType::paged;

    // Allocate Paged Memory as no type was specified.
    createPagedMemory();

    this->M[0] = 0;
}

template<typename type>
Propulsion::Matrix<type>::Matrix(const Matrix<type>& copyM) : M(nullptr, freePagedMemory)
{
    this->rows = copyM.rows;
    this->cols = copyM.cols;
    this->totalSize = copyM.totalSize;
    this->memoryType = copyM.memoryType;

    if(this->memoryType == Matrix < type > ::MatrixMemType::pinned) {
        createPinnedMemory();
    }
    else if(this->memoryType == Matrix < type > ::MatrixMemType::paged){
        createPagedMemory();
    }

    for(unsigned i = 0; i < this->totalSize; i++)
    {
        this->M[i] = copyM.M[i];
    }
}


template<typename type>
Propulsion::Matrix<type>::Matrix(const Matrix<type>& copyM, MatrixMemType memType) : M(nullptr, freePagedMemory)
{
    this->rows = copyM.rows;
    this->cols = copyM.cols;
    this->totalSize = copyM.totalSize;
    this->memoryType = memType;

    if(this->memoryType == Matrix < type > ::MatrixMemType::pinned) {
        createPinnedMemory();
    }
    else if(this->memoryType == Matrix < type > ::MatrixMemType::paged){
        createPagedMemory();
    }

    for(unsigned i = 0; i < this->totalSize; i++)
    {
        this->M[i] = copyM.M[i];
    }
}

template<typename type>
Propulsion::Matrix<type>::Matrix(Matrix<type>&& copyM) : M(nullptr, freePagedMemory)
{
    this->rows = copyM.rows;
    this->cols = copyM.cols;
    this->totalSize = copyM.totalSize;
    this->memoryType = copyM.memoryType;
    this->M = std::move(copyM.M);
}



template<typename type>
Propulsion::Matrix<type>::Matrix(unsigned rowAndColSize, Propulsion::Matrix<type>::MatrixMemType memType, MatrixInitVal miv, type customVal, MatrixInitType mit ) : M(nullptr, freePagedMemory)
{
    if(rowAndColSize > 0)
    {
        this->rows = rowAndColSize;
        this->cols = rowAndColSize;
        this->totalSize = rowAndColSize*rowAndColSize;
        this->memoryType = memType;

        generateMatrix(miv, mit, customVal, nullptr);
    }
}

template<typename type>
Propulsion::Matrix<type>::Matrix(unsigned rows, unsigned cols, Propulsion::Matrix<type>::MatrixMemType memType, MatrixInitVal miv, type customVal, MatrixInitType mit) : M(nullptr, freePagedMemory)
{
    if(rows > 0 && cols > 0)
    {
        this->rows = rows;
        this->cols = cols;
        this->totalSize = rows*cols;
        this->memoryType = memType;

        generateMatrix(miv, mit, customVal, nullptr);
    }
}

template <typename type>
Propulsion::Matrix<type>::Matrix(type *array, unsigned rowAndColSize, Propulsion::Matrix<type>::MatrixMemType memType) : M(nullptr, freePagedMemory)
{
    if(rowAndColSize > 0)
    {
        this->rows = rowAndColSize;
        this->cols = rowAndColSize;
        this->totalSize = rowAndColSize*rowAndColSize;
        this->memoryType = memType;

        generateMatrix(MatrixInitVal::custom, MatrixInitType::def, NULL, array);
    }
}

template<typename type>
Propulsion::Matrix<type>::Matrix(type *array, unsigned rows, unsigned cols, Propulsion::Matrix<type>::MatrixMemType memType,MatrixInitVal miv, type customVal, MatrixInitType mit) : M(nullptr, freePagedMemory)
{
    if(rows > 0 && cols > 0)
    {
        this->rows = rows;
        this->cols = cols;
        this->totalSize = rows*cols;
        this->memoryType = memType;

        generateMatrix(miv, mit, customVal, array);
    }
}

template<typename type>
Propulsion::Matrix<type>::~Matrix()
{ }

template<typename type>
void Propulsion::Matrix<type>::print(std::ostream& ostream)
{
    unsigned spaceCount = 0;
    std::string digitStr;
    for(unsigned i = 0; i < rows*cols; i++)
    {
        std::string digitStr = std::to_string(M[i]);
        if(digitStr.length() > spaceCount)
        {
            spaceCount = digitStr.length();
        }
        digitStr.clear();
    }


    for(unsigned i = 0; i < rows; i++)
    {
        ostream << "|";
        for(unsigned j = 0; j < cols; j++)
        {
            ostream << std::setw(spaceCount + 2) << M[i*cols + j];
        }
        ostream << " |" << std::endl;
    }
}


template<typename type>
void Propulsion::Matrix<type>::print(type *a, unsigned rows, unsigned cols)
{
    unsigned spaceCount = 0;
    std::string digitStr;
    for(unsigned i = 0; i < rows * cols; i++)
    {
        std::string digitStr = std::to_string(a[i]);
        if(digitStr.length() > spaceCount)
        {
            spaceCount = digitStr.length();
        }
    }


    for(unsigned i = 0; i < rows; i++)
    {
        std::cout << "|";
        for(unsigned j = 0; j < cols; j++)
        {
            std::cout << std::setw(spaceCount + 2) << a[i*rows + j];
        }
        std::cout << " |" << std::endl;
    }
}

template <typename type>
std::unique_ptr<type[], void(*)(type*)>  Propulsion::Matrix<type>::generateNullMatrixArray(unsigned rowAndColSize)
{
    // Get size of matrix.
    unsigned sz = rowAndColSize;
    std::unique_ptr<type[], void(*)(type*)> r(nullptr, freePagedMemory);

    if(this->memoryType == Matrix<type>::MatrixMemType::paged) {
        type* _pagedArr;
        _pagedArr = new type[sz];
        r = std::unique_ptr<type[], void(*)(type*)>(_pagedArr, freePagedMemory);
    }
    else if(this->memoryType == Matrix<type>::MatrixMemType::pinned) {
        type* _pinnedArr;
        gpuErrchk(hipHostMalloc((void**)&_pinnedArr, sz * sizeof(type)));
        r = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);
    }

    for(unsigned i = 0; i < sz; i++)
    {
        r[i] = NULL;
    }
    return r;
}

template <typename type>
std::unique_ptr<type[], void(*)(type*)>  Propulsion::Matrix<type>::generateZeroMatrixArray(unsigned rowAndColSize)
{
    // Why did I do this???
    unsigned sz = rowAndColSize;
    std::unique_ptr<type[], void(*)(type*)> r(nullptr, freePagedMemory);

    if(this->memoryType == Matrix<type>::MatrixMemType::paged) {
        type* _pagedArr;
        _pagedArr = new type[sz];
        r = std::unique_ptr<type[], void(*)(type*)>(_pagedArr, freePagedMemory);
    }
    else if(this->memoryType == Matrix<type>::MatrixMemType::pinned) {
        type* _pinnedArr;
        gpuErrchk(hipHostMalloc((void**)&_pinnedArr, sz * sizeof(type)));
        r = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);
    }


    for(unsigned i = 0; i < sz; i++)
    {
        r[i] = (type)0;
    }
    return r;
}

template<typename type>
std::unique_ptr<type[], void(*)(type*)>  Propulsion::Matrix<type>::generateDefaultMatrixArray(unsigned rowAndColSize, type customVal, type *array)
{
    unsigned sz = rowAndColSize;
    std::unique_ptr<type[], void(*)(type*)> r(nullptr, freePagedMemory);

    if(this->memoryType == Matrix<type>::MatrixMemType::paged) {
        type* _pagedArr;
        _pagedArr = new type[sz];
        r = std::unique_ptr<type[], void(*)(type*)>(_pagedArr, freePagedMemory);
    }
    else if(this->memoryType == Matrix<type>::MatrixMemType::pinned) {
        type* _pinnedArr;
        gpuErrchk(hipHostMalloc((void**)&_pinnedArr, sz * sizeof(type)));
        r = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);
    }

    if(array == nullptr)
    {
        for(unsigned i = 0; i < rowAndColSize; i++)
        {
            r[i] =customVal;
        }
    }
    else
    {
        for(unsigned i = 0; i < rowAndColSize; i++)
        {
            r[i] = array[i];
        }
    }

    return r;
}

template <typename type>
std::unique_ptr<type[], void(*)(type*)> Propulsion::Matrix<type>::generateDiagonalMatrixArray(unsigned rowAndColSize, type customVal, type *array)
{
    unsigned sz = rowAndColSize;
    std::unique_ptr<type[], void(*)(type*)> r(nullptr, freePagedMemory);

    if(this->memoryType == Matrix<type>::MatrixMemType::paged) {
        type* _pagedArr;
        _pagedArr = new type[sz];
        r = std::unique_ptr<type[], void(*)(type*)>(_pagedArr, freePagedMemory);
    }
    else if(this->memoryType == Matrix<type>::MatrixMemType::pinned) {
        type* _pinnedArr;
        gpuErrchk(hipHostMalloc((void**)&_pinnedArr, sz * sizeof(type)));
        r = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);
    }

    if(array == nullptr)
    {
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                if(i == j)
                {
                    r[i*this->cols + j] = customVal;
                }
                else
                {

                    r[i*this->cols + j] = NULL;
                }
            }
        }
    }
    else
    {
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                if(i == j)
                {
                    r[i*this->cols + j] = array[i];
                }
                else
                {
                    r[i*this->cols + j] = NULL;
                }
            }
        }
    }
    return r;
}

template <typename type>
void Propulsion::Matrix<type>::T()
{
    // Generate new array as we need a new one.
    Propulsion::Matrix<type> temp(this->rows, this->cols, this->memoryType);

    for(unsigned i = 0; i < this->rows; i++)
    {
        for(unsigned j = 0; j < this->cols; j++)
        {
            temp.M[j*this->rows + i] = this->M[i*this->cols + j];
        }
    }

    // Alter the rows and cols to their new representation.
    unsigned t = this->rows;
    this->rows = this->cols;
    this->cols = t;

    this->M = std::move(temp.M);
}

template <typename type>
void Propulsion::Matrix<type>::add(const Matrix<type>& B, bool printTime)
{
    if(B.rows == this->rows && B.cols == this->cols)
    {
        Propulsion::Matrix<type> temp(this->rows, this->cols, this->memoryType);

        //auto temp = std::make_unique<type[]>(this->rows * this->cols);
        // Use CUDA to speed up the process.
        if(this->totalSize >= MATRIX_CUDA_ADD_DIFF_ELEM_SIZE)
        {
            cudaAdd1DArraysWithStride(this->M.get(), B.M.get(), temp.M.get(), this->totalSize, printTime);
        }
        // Else just do it via HOST avx.
        else
        {
            Propulsion::hostAdd1DArraysAVX256(this->M.get(), B.M.get(), temp.M.get(), this->totalSize, printTime);
        }

        this->M = std::move(temp.M);
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                                                        __FILE__, __LINE__, "add" , "Addition Requires all dimension sizes to be the same as the operation is element wise.");
    }
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::addRowVector(Matrix<type> &b)
{
    // Create return matrix. Initialized as 1x1 zero matrix.
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);

    // Check if the rowVector can even be added to this. And check if b is a vector.
    if(this->cols == b.cols && b.rows == 1)
    {
        // Loop through every element of this, add the jth element from be to every (i,j) of this.
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                ret(i,j) = this->at(i,j) + b(j);
            }
        }
    }
    else
    {
        // Error for add row vector
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) +
                          ") vs. (" + std::to_string(b.rows) + ", " + std::to_string(b.cols) + ")" + ". Expected second Matrix to be ( 1, " + std::to_string(b.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),__FILE__, __LINE__, "addRowVector" ,
                                                        "addRowVector Requires that the argument be a row vector such that it is 1xn in DIM.");
    }
    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::addRowVector(Matrix<type> &&b)
{
    // Create return matrix. Initialized as 1x1 zero matrix.
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);

    // Check if the rowVector can even be added to this. And check if b is a vector.
    if(this->cols == b.cols && b.rows == 1)
    {
        // Set the return matrix to the size of this.
        /*
        ret.rows = this->rows;
        ret.cols = this->cols;
        ret.totalSize = this->totalSize;
        ret.M = std::make_unique<type[]>(this->totalSize);*/

        // Loop through every element of this, add the jth element from be to every (i,j) of this.
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                ret(i,j) = this->at(i,j) + b(j);
            }
        }
    }
    else
    {
        // Error for add row vector
        std::string err = "Matrix Size Mismatch, ("+ std::to_string(this->rows) + ", " + std::to_string(this->cols)  +
                          ") vs. (" + std::to_string(b.rows) +", " + std::to_string(b.cols) + ")" + ". Expected second Matrix to be ( 1, " + std::to_string(b.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),__FILE__, __LINE__, "addRowVector" ,
                                                        "addRowVector Requires that the argument be a row vector such that it is 1xn in DIM.");
    }
    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::addColVector(Matrix<type> &b)
{
    // Create return matrix. Initialized as 1x1 zero matrix.
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);

    // Check if the colVector can even be added to the matrix.
    if(this->rows == b.rows && b.cols == 1)
    {
        // Set the return matrix to the size of this.
        /*
        ret.rows = this->rows;
        ret.cols = this->cols;
        ret.totalSize = this->totalSize;
        ret.M = std::make_unique<type[]>(this->totalSize);*/

        // Loop through every element of this, add the ith element from be to every (i,j) of this.
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                ret(i,j) = this->at(i,j) + b(i);
            }
        }
    }
    else
    {
        // Error for add row vector
        std::string err = "Matrix Size Mismatch, ("+ std::to_string(this->rows) + ", " + std::to_string(this->cols)  +
                          ") vs. (" + std::to_string(b.rows) +", " + std::to_string(b.cols) + ")" + ". Expected second Matrix to be ( " + std::to_string(b.rows) + ", 1)";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),__FILE__, __LINE__, "addColVector" ,
                                                        "addColVector Requires that the argument be a row vector such that it is nx1 in DIM.");
    }
    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::addColVector(Matrix<type> &&b)
{
    // Create return matrix. Initialized as 1x1 zero matrix.
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);

    // Check if the colVector can even be added to the matrix.
    if(this->rows == b.rows && b.cols == 1)
    {
        // Set the return matrix to the size of this.
        /*
        ret.rows = this->rows;
        ret.cols = this->cols;
        ret.totalSize = this->totalSize;
        ret.M = std::make_unique<type[]>(this->totalSize);*/

        // Loop through every element of this, add the ith element from be to every (i,j) of this.
        for(unsigned i = 0; i < this->rows; i++)
        {
            for(unsigned j = 0; j < this->cols; j++)
            {
                ret(i,j) = this->at(i,j) + b(i);
            }
        }
    }
    else
    {
        // Error for add row vector
        std::string err = "Matrix Size Mismatch, ("+ std::to_string(this->rows) + ", " + std::to_string(this->cols)  +
                          ") vs. (" + std::to_string(b.rows) +", " + std::to_string(b.cols) + ")" + ". Expected second Matrix to be ( " + std::to_string(b.rows) + ", 1)";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),__FILE__, __LINE__, "addColVector" ,
                                                        "addColVector Requires that the argument be a row vector such that it is nx1 in DIM.");
    }
    return ret;
}

template <typename type>
void Propulsion::Matrix<type>::subtract(const Matrix<type> &B, bool printTime)
{
    if(B.rows == this->rows && B.cols == this->cols)
    {
        Propulsion::Matrix<type> temp(this->rows, this->cols, this->memoryType);
        // Use CUDA to speed up the process.
        if(this->totalSize >= MATRIX_CUDA_ADD_DIFF_ELEM_SIZE)
        {
            cudaSubtract1DArraysWithStride(this->M.get(), B.M.get(), temp.M.get(), this->rows * this->cols, printTime);
        }
            // Else just do it via HOST avx.
        else
        {
            Propulsion::hostSubtract1DArraysAVX256(this->M.get(), B.M.get(), temp.M.get(), this->totalSize, printTime);
        }

        this->M = std::move(temp.M);
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                                                        __FILE__, __LINE__, "subtract" , "Subtraction Requires all dimension sizes to be the same as the operation is element wise.");
    }
}

template<typename type>
void Propulsion::Matrix<type>::cudaDotProduct(const Matrix<type> &B, bool printTime)
{
    if(this->cols == B.rows) {
        // Create Matrix with A row size and b col size as nxm * mxk = nxk
        Propulsion::Matrix<type> temp(this->getRowSize(), B.cols, this->memoryType, MatrixInitVal::zero);

        // Using CUDA from Propulsion to handle.
        Propulsion::cudaDotProduct(this->getArray(), B.M.get(), temp.getArray(), this->getRowSize(), this->cols,
                                   B.cols, printTime);

        // Move the pointer from Temp to M now.
        this->M = std::move(temp.M);
        this->rows = temp.rows;
        this->cols = temp.cols;
        this->totalSize = temp.totalSize;
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                                                        __FILE__, __LINE__, "cudaMultiplyMatrices" , "cudaMultiplyMatrices Requires the second dimension of matrix A match first dimension of matrix B");
    }
}

template <typename type>
void Propulsion::Matrix<type>::dot(const Matrix<type> &B, bool printTime)
{
    if(this->cols == B.rows)
    {
        // Get size of new Matrix
        unsigned newSize = this->rows * B.cols;
        Propulsion::Matrix<type> multiplyArray(this->rows, B.cols, this->memoryType);

        // Case its a 1x1.
        if(this->totalSize == 1 && B.totalSize == 1)
        {
            multiplyArray.M[0] = this->M[0] * B.M[0];
        }
        else {
            /*
            for (unsigned r = 0; r < this->rows; r++) {
            for (unsigned c = 0; c < b.cols; c++) {
            for (unsigned i = 0; i < this->cols; i++) {
            sum += at(r, i) * b.M[i * b.cols + c];
            }
            multiplyArray[n] = sum;
            sum = 0;
            n++;
            }
            }*/

            Propulsion::hostDotProduct(this->M.get(), B.M.get(), multiplyArray.M.get(), this->rows, this->cols, B.cols, printTime);
        }

        this->cols = B.cols;        // If you know, you know. AB: A * B is 2x3 - 3x3: New Matrix is 2(this rows)x3(b cols).
        this->totalSize = newSize;  // set the totalSize of this to the new size of the product matrix.
        this->M = std::move(multiplyArray.M);
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                                                        __FILE__, __LINE__, "dot" , "dot Requires the second dimension of matrix A match first dimension of matrix B");
    }
}

template<typename type>
void Propulsion::Matrix<type>::schurProduct(const Matrix<type> &B, bool printTime)
{
    if(this->rows == B.rows && this->cols == B.cols)
    {
        Propulsion::Matrix<type> schurArray(this->rows, this->cols, this->memoryType);

        if (this->totalSize >= MATRIX_CUDA_ADD_DIFF_ELEM_SIZE)
        {
            cudaSchurProduct(this->M.get(), B.M.get(), schurArray.M.get(), this->totalSize, printTime);
        }
        else
        {
            hostSchurProduct(this->M.get(), B.M.get(), schurArray.M.get(), this->totalSize, printTime);
        }

        this->M = std::move(schurArray.M);
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                __FILE__, __LINE__, "schurProduct" , "Schurs Product Requires all dimension sizes to be the same, as the proudct is element wise.");
    }
}

template<typename type>
void Propulsion::Matrix<type>::multiply(type scalar) noexcept
{
    for(unsigned i = 0; i < this->totalSize; i++)
    {
        this->M[i] *= scalar;
    }
}

template<typename type>
void Propulsion::Matrix<type>::strassenMultiplication(const Matrix<type> &B)
{
    // check if we can dot first.
    if(this->cols == B.rows)
    {
        // get the log2(rows/cols) as we need to make a nxn matrix that is divisible into 4 partitions.
        double firstRowPowerOfTwo  = std::log2(this->rows);
        double firstColPowerOfTwo  = std::log2(this->cols); // This value is the same as b.cols, so we only need to check the Column.
        double secondColPowerOfTwo = std::log2(B.cols);

        double largestNPower = 0.0;

        // then find the largest value of the 3, store into largestNPower, so we can pad later with zeroes to the ceiling
        // of largestNPower.
        if(firstRowPowerOfTwo >= firstColPowerOfTwo)
        {
            largestNPower = firstRowPowerOfTwo;
        }
        else
        {
            largestNPower = firstColPowerOfTwo;
        }
        if(largestNPower <= secondColPowerOfTwo)
        {
            largestNPower = secondColPowerOfTwo;
        }

        // ceiling to the next 2^largestNPower.
        unsigned squareR = (unsigned)(std::pow(2.0, std::ceil(largestNPower)));

        // the rows and cols of the matrices are now altered if they need to be.
        Propulsion::Matrix<type> A = *this;
        Propulsion::Matrix<type> B = B;

        // pad the matrices with zeros for Strassen Multiplication.
        A.pad(squareR, squareR);
        B.pad(squareR, squareR);

        // Divide A and B into 8 partitions starting with 4 from A.
        auto a = A.getRangeMatrix(0, A.rows / 2 - 1, 0, A.cols / 2 - 1);
        auto b = A.getRangeMatrix(0, A.rows / 2 - 1, A.cols / 2, A.cols - 1);
        auto c = A.getRangeMatrix(A.rows / 2, A.rows - 1, 0, A.cols / 2 - 1);
        auto d = A.getRangeMatrix(A.rows / 2, A.rows - 1, A.cols / 2, A.cols - 1);
        auto e = B.getRangeMatrix(0, B.rows / 2 - 1, 0, B.cols / 2 - 1);
        auto f = B.getRangeMatrix(0, B.rows / 2 - 1, B.cols / 2, B.cols - 1);
        auto g = B.getRangeMatrix(B.rows / 2, B.rows - 1, 0, B.cols / 2 - 1);
        auto h = B.getRangeMatrix(B.rows / 2, B.rows - 1, B.cols / 2, B.cols - 1);

        // Create 7 async threads for the Strassen recursion.
        auto p1 = std::async(recursiveStrassen, a, f - h);
        auto p2 = std::async(recursiveStrassen, a + b, h);
        auto p3 = std::async(recursiveStrassen, c + d, e);
        auto p4 = std::async(recursiveStrassen, d, g - e);
        auto p5 = std::async(recursiveStrassen, a + d, e + h);
        auto p6 = std::async(recursiveStrassen, b - d, g + h);
        auto p7 = std::async(recursiveStrassen, a - c, e + f);

        // Get Values from threads for future use.
        auto pr1 = p1.get();
        auto pr2 = p2.get();
        auto pr3 = p3.get();
        auto pr4 = p4.get();
        auto pr5 = p5.get();
        auto pr6 = p6.get();
        auto pr7 = p7.get();

        auto c1 = pr5 + pr4 - pr2 + pr6;
        auto c2 = pr1 + pr2;
        auto c3 = pr3 + pr4;
        auto c4 = pr1 + pr5 - pr3 - pr7;

        auto C = c1.mergeRight(c2);
        auto CB = c3.mergeRight(c4);
        C = C.mergeBelow(CB);

        // trim the matrix to the original pxn*mxq = pxq
        for(unsigned i = 0; i < squareR - B.cols; i++)
        {
            C = C.removeCol(b.cols);    // Remove Last col i times.
        }

        for(unsigned i = 0; i < squareR - this->rows; i++)
        {
            C = C.removeRow(this->rows);    // Remove Last row i times.
        }

        *this = C;
    }
    else
    {
        std::string err = "Matrix Size Mismatch, (" + std::to_string(this->rows) + ", " + std::to_string(this->cols) + ") vs. (" + std::to_string(B.rows) + ", " + std::to_string(B.cols) + ")";
        throw Propulsion::Matrix<type>::MatrixException(err.c_str(),
                                                        __FILE__, __LINE__, "dot" , "dot Requires the second dimension of matrix A match first dimension of matrix B");
    }
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::recursiveStrassen(Propulsion::Matrix<type> A, Propulsion::Matrix<type> B) {
    // check the size of the leaf matrix. That way we can just use O(n^3)
    // once we get to a manageable size.
    if(A.totalSize <= HOST_STRASSEN_LEAF_SIZE)
    {
        Propulsion::Matrix<type> ret = A;
        ret.dot(B);
        return ret;
    }

    // get range matrices for a,b,c,d,e,f,g and h for strassen multiplication. You know you know.
    auto a = A.getRangeMatrix(0, A.rows / 2 - 1, 0, A.cols / 2 - 1);
    auto b = A.getRangeMatrix(0, A.rows / 2 - 1, A.cols / 2, A.cols - 1);
    auto c = A.getRangeMatrix(A.rows / 2, A.rows - 1, 0, A.cols / 2 - 1);
    auto d = A.getRangeMatrix(A.rows / 2, A.rows - 1, A.cols / 2, A.cols - 1);
    auto e = B.getRangeMatrix(0, B.rows / 2 - 1, 0, B.cols / 2 - 1);
    auto f = B.getRangeMatrix(0, B.rows / 2 - 1, B.cols / 2, B.cols - 1);
    auto g = B.getRangeMatrix(B.rows / 2, B.rows - 1, 0, B.cols / 2 - 1);
    auto h = B.getRangeMatrix(B.rows / 2, B.rows - 1, B.cols / 2, B.cols - 1);

    auto p1 = recursiveStrassen(a, f - h);
    auto p2 = recursiveStrassen(a + b, h);
    auto p3 = recursiveStrassen(c + d, e);
    auto p4 = recursiveStrassen(d, g - e);
    auto p5 = recursiveStrassen(a + d, e + h);
    auto p6 = recursiveStrassen(b - d, g + h);
    auto p7 = recursiveStrassen(a - c, e + f);

    auto c1 = p5 + p4 - p2 + p6;
    auto c2 = p1 + p2;
    auto c3 = p3 + p4;
    auto c4 = p1 + p5 - p3 - p7;

    auto C = c1.mergeRight(c2);
    auto CB = c3.mergeRight(c4);
    C = C.mergeBelow(CB);


    return C;
}

template<typename type>
type* Propulsion::Matrix<type>::getArray()
{
    return this->M.get();
}

template<typename type>
unsigned Propulsion::Matrix<type>::getColSize()
{
    return cols;
}

template<typename type>
unsigned Propulsion::Matrix<type>::getRowSize()
{
    return rows;
}

template<typename type>
unsigned Propulsion::Matrix<type>::getTotalSize()
{
    return cols * rows;
}

template <typename type>
bool Propulsion::Matrix<type>::equalTo(const Matrix<type> &B)
{
    if(this->rows != B.rows || this->cols != B.cols)
    {
        return false;
    }
    else
    {
        for(unsigned i = 0; i < this->totalSize; i++)
        {
            if(this->M[i] != B.M[i])
            {
                return false;
            }
        }
        return true;
    }
}

template<typename type>
bool Propulsion::Matrix<type>::operator==(const Propulsion::Matrix<type> &rhs) {
    return equalTo(rhs);
}

template<typename type>
bool Propulsion::Matrix<type>::isUpperTriangular()
{
    // Not nxn.
    if(this->rows != this->cols){return false;}

    for(unsigned i = 0; i < this->rows; i++)
    {
        for(unsigned j = 0; j <= i; j++)
        {
            // On the diagonal.
            if(i == j)
            {
                // if diagonal is nonzero, move on.
                if(at(i,j) != 0)
                {
                    continue;
                }
                else
                {
                    return false;
                }
            }

            if(at(i,j) == 0)
            {
                continue;
            }
            else
            {
                return false;
            }
        }
    }
    return true;
}

template<typename type>
bool Propulsion::Matrix<type>::isLowerTriangular()
{
    // Not nxn.
    if(this->rows != this->cols){return false;}

    for(unsigned i = 0; i < this->rows; i++)
    {
        for(unsigned j = i; j < this->cols; j++)
        {
            // On the diagonal.
            if(i == j)
            {
                if(at(i,j) != 0)
                {
                    continue;
                }
                else
                {
                    return false;
                }
            }
            if(at(i,j) == 0)
            {
                continue;
            }
            else
            {
                return false;
            }
        }
    }
    return true;
}


template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::getRowMatrix(unsigned int row)
{
    Propulsion::Matrix<type> ret(1, this->cols, this->memoryType);
    // Check if in range.
    if(row < this->rows) {
        for (unsigned i = 0; i < ret.cols; i++) {
            ret.M[i] = this->M[row * this->cols + i];
        }
    }

    return ret;
}

template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::getColMatrix(unsigned int col)
{
    Propulsion::Matrix<type> ret(this->rows, 1, this->memoryType);
    // Check if in range.
    if(col < this->cols) {
        for (unsigned i = 0; i < ret.rows; i++) {
            ret.M[i] = this->M[this->cols * i + col];
        }
    }

    return ret;
}

template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::getRangeMatrix(unsigned rowStart, unsigned rowEnd, unsigned colStart, unsigned colEnd)
{
    Propulsion::Matrix<type> ret(rowEnd - rowStart + 1, colEnd - colStart + 1, this->memoryType);
    // Check if in range.
    if(rowStart <= rowEnd && rowEnd < this->rows && colStart <= colEnd && colEnd < this->cols)
    {
        unsigned rElement = 0; // Iterator for the nth element in the return array. Incremented in second for loop.


        for(unsigned i = rowStart; i <= rowEnd; i++)
        {
            for(unsigned j = colStart; j <= colEnd; j++)
            {
                ret.M[rElement] = at(i,j);
                rElement++;
            }
        }
    }

    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::mergeRight( Matrix<type> &B)
{
    Propulsion::Matrix<type> ret(this->rows, this->cols + B.cols, this->memoryType);
    // Check whether or not they have the same rows.
    if(this->rows == B.rows)
    {
        for(unsigned i = 0; i < ret.rows; i++)
        {
            for(unsigned j = 0; j < ret.cols; j++)
            {
                if(j < this->cols)
                {
                    ret.at(i,j) = at(i,j);
                }
                else
                {
                    ret.at(i,j) = B.at(i, j - this->cols);
                }
            }
        }
        return ret;
    }

    return *this;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::mergeBelow( Matrix<type> &B)
{
    Propulsion::Matrix<type> ret(this->rows + B.rows, this->cols, this->memoryType);
    // Check whether or not they have the same rows.
    if(this->cols == B.cols)
    {
        for(unsigned i = 0; i < ret.rows; i++)
        {
            for(unsigned j = 0; j < ret.cols; j++)
            {
                if(i < this->rows)
                {
                    ret.at(i,j) = at(i,j);
                }
                else
                {
                    ret.at(i,j) = B.at(i - this->rows, j);
                }
            }
        }
        return ret;
    }

    // return the object called from which is unaltered.
    return *this;
}

template <typename type>
type& Propulsion::Matrix<type>::at(unsigned i)
{
    if(i < rows * cols)
        return this->M[i];
    else {
        /*std::cout << "Accessing Matrice Outside of Bounds with i: " << i
                  << ", DIMS is [" << rows << "," << cols << "] = " << rows * cols << std::endl;*/
        throw std::out_of_range("Accessing Matrix Outside of Bounds with i: " + std::to_string(i) + ", DIMS is [" + std::to_string(rows)
                                + "," + std::to_string(cols) + "] = " + std::to_string(rows*cols) + "\n");
    }
}

template <typename type>
type& Propulsion::Matrix<type>::at(unsigned i, unsigned j)
{
    if(i < rows && j < cols)
    {
        return this->M[i*cols + j];
    }
    else
    {
        throw std::out_of_range("Accessing Matrix Outside of Bounds with i: " + std::to_string(i) + " & j: " + std::to_string(j) + ", DIMS is ["
                                + std::to_string(rows) + "," + std::to_string(cols) + "] = " + std::to_string(rows*cols) + "\n");
    }
}

template <typename type>
type& Propulsion::Matrix<type>::operator()(unsigned i)
{
    return at(i);
}

template <typename type>
type& Propulsion::Matrix<type>::operator()(unsigned i, unsigned j)
{
    return at(i,j);
}


template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::operator+(Matrix<type> &rhs)
{
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);
    if(rows == rhs.rows && cols == rhs.cols)
    {
        hostAdd1DArraysAVX256(this->getArray(), rhs.getArray(), ret.getArray(),this->getTotalSize());
        /*
        for(unsigned i = 0; i < totalSize; i++)
        {
            ret.M[i] = rhs.M[i] + M[i];
        }*/
    }
    return ret;
}

template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::operator-(Matrix<type> &rhs)
{
    Propulsion::Matrix<type> ret(this->rows, this->cols, this->memoryType);
    if(rows == rhs.rows && cols == rhs.cols)
    {
        hostSubtract1DArraysAVX256(this->getArray(), rhs.getArray(), ret.getArray(), this->getTotalSize());
        /*
        for(unsigned i = 0; i < totalSize; i++)
        {
            ret.M[i] = M[i] - rhs.M[i];
        }*/
    }
    return ret;
}

template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::operator*(const Matrix<type> &rhs) {
    Propulsion::Matrix<type> ret = *this;    // Copy the contents of this to the return value.


    if(ret.totalSize < MATRIX_CUDA_DOT_ELEM_SIZE)
    {
        ret.dot(rhs);      // Use already defined object method to dot by the right hand side matrix.
    }
    else
    {
        ret.cudaDotProduct(rhs);
    }

    return ret;
}

template <typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::operator*(type rhs)
{
    Propulsion::Matrix<type> ret = *this;    // Copy the contents of this to the return value.
    ret.multiply(rhs);      // Use already defined object method to dot by the right hand side scalar.
    return ret;
}


template<typename type>
Propulsion::Matrix<type>& Propulsion::Matrix<type>::operator=(const Matrix<type> &rhs)
{
    // If same return this
    if(this == &rhs) {return *this;}
    else {
        // else, make a deep copy.
        this->totalSize = rhs.totalSize;
        this->rows = rhs.rows;
        this->cols = rhs.cols;

        type* _pinnedArr;
        gpuErrchk(hipHostMalloc((void**)&_pinnedArr, this->totalSize * sizeof(type)));
        this->M = std::unique_ptr<type[], void(*)(type*)>(_pinnedArr, freePinnedMemory);

        for (unsigned i = 0; i < rhs.totalSize; i++) {
            this->M[i] = rhs.M[i];
        }
    }
    return *this;
}

/*
template<typename type>
Propulsion::Matrix<type>& Propulsion::Matrix<type>::operator=(Matrix<type> &r)
{
    if(this == &r) {return *this;}
    else {
        delete[] this->M;
        this->M = new type[r.totalSize];
        this->rows = r.rows;
        this->cols = r.cols;
        this->totalSize = r.totalSize;

        for (unsigned i = 0; i < r.totalSize; i++) {
            this->M[i] = r.M[i];
        }
    }
    return *this;
}*/

template <typename type>
void Propulsion::Matrix<type>::pad(unsigned rows, unsigned cols)
{
    if(this->rows <= rows && this->cols <= cols)
    {
        // the number of rows/cols to add. E.g 6-4=2 rows to add.
        unsigned rowsToAdd = rows - this->rows;
        unsigned colsToAdd = cols - this->cols;


        Propulsion::Matrix<type> bottomRows(rowsToAdd, this->cols, this->memoryType);
        Propulsion::Matrix<type> rightCols(rows, colsToAdd, this->memoryType);


        if(this->rows < rows)
            *this = mergeBelow(bottomRows);
        if(this->cols < cols)
            *this = mergeRight(rightCols);
    }
}

template<typename type>
void Propulsion::Matrix<type>::populateWithUniformDistribution(type lRange, type rRange)
{
    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution((double)lRange, (double)rRange);

    for(unsigned i = 0; i < this->totalSize;i++)
    {
        this->M[i] = (type)distribution(generator);
    }
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::removeRow(unsigned int rowToRem) {
    Propulsion::Matrix<type> ret(this->rows - 1, this->cols, this->memoryType);
    unsigned rIter = 0;
    if(rowToRem < this->rows)
    {
        for(unsigned i = 0; i < ret.rows; i++)
        {
            if(i == rowToRem){rIter++;}
            for(unsigned j = 0; j < ret.cols; j++)
            {
                ret.at(i,j) = at(rIter,j);
            }

            rIter++;
        }
        return ret;
    }

    return *this;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::removeCol(unsigned int colToRem) {
    Propulsion::Matrix<type> ret(this->rows, this->cols - 1, this->memoryType);
    unsigned cIter = 0;
    if(colToRem < this->cols)
    {
        for(unsigned i = 0; i < ret.rows; i++)
        {
            cIter = 0;
            for(unsigned j = 0; j < ret.cols; j++)
            {
                if(j == colToRem){cIter++;}
                ret.at(i,j) = at(i,cIter);
                cIter++;
            }
        }
        return ret;
    }

    return *this;
}



template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::copy(Matrix<type> copyM)
{
    Propulsion::Matrix<type> b(copyM.getRowSize(), copyM.getColSize(), copyM.memoryType);

    if(copyM.totalSize > MATRIX_COPY_SIZE_DIFF) {
        Propulsion::cudaCopyArray(copyM.M.get(), b.M.get(), copyM.getTotalSize());
        return b;
    }
    else
    {
        b = copyM;
        return b;
    }
}


template<typename type>
void Propulsion::Matrix<type>::randomRealDistribution(Matrix<type> &A, type lVal, type rVal)
{
    std::random_device rd;
    std::mt19937 e2(rd());
    std::uniform_real_distribution<> dist(lVal,rVal);

    for(unsigned i = 0; i < A.getTotalSize(); i++)
    {
        A(i) = dist(e2);
    }
}

template<typename type>
void Propulsion::Matrix<type>::randomRealDistribution(std::shared_ptr<Matrix<type>> A, type lVal, type rVal)
{
    std::random_device rd;
    std::mt19937 e2(rd());
    std::uniform_real_distribution<> dist(lVal,rVal);

    for(unsigned i = 0; i < A->getTotalSize(); i++)
    {
        A->at(i) = dist(e2);
    }
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::sumRows(Matrix<type> &&A)
{
    // create return object, give it size of the rows from A, 1 for columns.
    Propulsion::Matrix<type> ret(A.rows, 1, A.memoryType);

    for(unsigned i = 0; i < A.rows; i++)
    {
        // Sum starts from zero.
        type sum = (type)0;

        // Loop through all of A, adding the elements on the same row.
        for(unsigned j = 0; j < A.cols; j++)
        {
            // populate the sum var.
            sum += A.at(i,j);
        }

        // Return matrix is populated with sum at every i value.
        ret.at(i) = sum;
    }

    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::sumRows(Matrix<type> &A)
{
    // create return object, give it size of the rows from A, 1 for columns.
    Propulsion::Matrix<type> ret(A.rows, 1, A.memoryType);

    for(unsigned i = 0; i < A.rows; i++)
    {
        // Sum starts from zero.
        type sum = (type)0;

        // Loop through all of A, adding the elements on the same row.
        for(unsigned j = 0; j < A.cols; j++)
        {
            // populate the sum var.
            sum += A.at(i,j);
        }

        // Return matrix is populated with sum at every i value.
        ret.at(i) = sum;
    }

    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::sumCols(Matrix<type> &&A)
{
    // create return object, give it size of the rows from A, 1 for rows.
    Propulsion::Matrix<type> ret(1, A.cols, A.memoryType);

    for(unsigned j = 0; j < A.cols; j++)
    {
        // Sum starts from zero.
        type sum = (type)0;

        // Loop through all of A, adding the elements on the same row.
        for(unsigned i = 0; i < A.rows; i++)
        {
            // populate the sum var.
            sum += A.at(i,j);
        }

        // Return matrix is populated with sum at every i value.
        ret.at(j) = sum;
    }

    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::sumCols(Matrix<type> &A)
{
    // create return object, give it size of the rows from A, 1 for columns.
    Propulsion::Matrix<type> ret(1, A.cols, A.memoryType);

    for(unsigned j = 0; j < A.cols; j++)
    {
        // Sum starts from zero.
        type sum = (type)0;

        // Loop through all of A, adding the elements on the same row.
        for(unsigned i = 0; i < A.rows; i++)
        {
            // populate the sum var.
            sum += A.at(i,j);
        }

        // Return matrix is populated with sum at every i value.
        ret.at(j) = sum;
    }

    return ret;
}

template<typename type>
type Propulsion::Matrix<type>::getMax()
{
    // The most unlikely event in this case.
    type max = M[0];

    for(unsigned i = 1; i < this->totalSize; i++)
    {
        // if M of i is greater than the current max value.
        if(M[i] > max)
        {
            max = M[i];
        }
    }

    return max;
}

template<typename type>
type Propulsion::Matrix<type>::getMin()
{
    // The most unlikely event in this case.
    type min = M[0];

    for(unsigned i = 1; i < this->totalSize; i++)
    {
        // If M of i is less than the current min value.
        if(M[i] < min)
        {
            min = M[i];
        }
    }

    return min;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::addBroadScalar(Matrix<type> &A, type s)
{
    Propulsion::Matrix<type> ret = A;

    for(unsigned i = 0; i < ret.totalSize; i++)
    {
        ret.at(i) += s;
    }

    return ret;
}

template<typename type>
Propulsion::Matrix<type> Propulsion::Matrix<type>::subtractBroadScalar(Matrix<type> &A, type s)
{
    Propulsion::Matrix<type> ret = A;

    for(unsigned i = 0; i < ret.totalSize; i++)
    {
        ret.at(i) -= s;
    }

    return ret;
}